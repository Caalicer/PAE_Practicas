#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 8192
#define BYTES SIZE * sizeof(int)

#define BLOCK_SIZE 256
#define GRID_SIZE (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE

// Error checking function
void checkCUDAError(const char* msg) {

	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {

		fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);

	}

}

// Kernel para inicializar el array
__global__ void initArray(int* x, int size) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {

		x[idx] = idx;

	}

}

// Kernel para invertir el array
__global__ void invertArray(int* x, int* y, int size) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {

		y[size - 1 - idx] = x[idx];

	}

}

// Function to verify result
bool verifyResult(int* y, int size) {

	for (int i = 0; i < size; i++) {

		if (y[i] != size - 1 - i) {

			printf("Error en la posición %d: esperado %d, obtenido %d\n", i, size - 1 - i, y[i]);
			return false;

		}

	}

	return true;

}

int main() {

	// Allocate host memory
	int* h_y = (int*)malloc(BYTES);

	// Allocate device memory
	int *d_x, *d_y;

	hipMalloc(&d_x, BYTES);
	hipMalloc(&d_y, BYTES);

	checkCUDAError("hipMalloc failed");

	// Initialize array on GPU
	initArray<<<GRID_SIZE, BLOCK_SIZE>>>(d_x, SIZE);
	hipDeviceSynchronize();

	checkCUDAError("Kernel initialization failed");

	// Create CUDA events for timing
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Record start time
	hipEventRecord(start);

	// Invert array
	invertArray<<<GRID_SIZE, BLOCK_SIZE>>>(d_x, d_y, SIZE);
	hipDeviceSynchronize();

	// Record stop time
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate elapsed time
	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Execution time: %.3f ms\n", milliseconds);

	// Check for errors
	checkCUDAError("Kernel execution failed");

	// Copy result back to host
	hipMemcpy(h_y, d_y, BYTES, hipMemcpyDeviceToHost);

	checkCUDAError("hipMemcpy failed");

	// Verify result

	if (verifyResult(h_y, SIZE)) {

		printf("Nice!\n");

	} else {

		printf("F!\n");

	}

	// Cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_x);
	hipFree(d_y);

	checkCUDAError("hipFree failed");

	free(h_y);

	return EXIT_SUCCESS;

}
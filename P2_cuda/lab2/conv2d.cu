#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include "pgmio.h"

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec / 1.0e9;
}

int conv2d_sequential(float* image, int img_w, int img_h, float* conv) {
    int ker_w = 3, ker_h = 3;
    float ker[] = {0, -1, 0, -1, 5, -1, 0, -1, 0};
    int pad_w = ker_w / 2;
    int pad_h = ker_h / 2;

    for (int y = 0; y < img_h; y++) {
        for (int x = 0; x < img_w; x++) {
            float sum = 0.0;
            for (int ky = 0; ky < ker_h; ky++) {
                for (int kx = 0; kx < ker_w; kx++) {
                    int img_x = x + kx - pad_w;
                    int img_y = y + ky - pad_h;
                    if (img_x >= 0 && img_x < img_w && img_y >= 0 &&
                        img_y < img_h) {
                        sum +=
                            image[img_y * img_w + img_x] * ker[ky * ker_w + kx];
                    }
                }
            }
            conv[y * img_w + x] = sum;
        }
    }
    return EXIT_SUCCESS;
}

void checkCUDAError(const char* msg) {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s in %s at line %d.\n", msg,
                hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

__global__ void conv2dKernel(float* image, float* conv, int img_w, int img_h,
                             float* ker, int ker_w, int ker_h) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < img_w && y < img_h) {
        int pad_w = ker_w / 2;
        int pad_h = ker_h / 2;
        float sum = 0.0f;

        for (int ky = 0; ky < ker_h; ky++) {
            for (int kx = 0; kx < ker_w; kx++) {
                int img_x = x + kx - pad_w;
                int img_y = y + ky - pad_h;
                if (img_x >= 0 && img_x < img_w && img_y >= 0 &&
                    img_y < img_h) {
                    sum += image[img_y * img_w + img_x] * ker[ky * ker_w + kx];
                }
            }
        }

        conv[y * img_w + x] = sum;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <image.pgm> <threadsPerBlock>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* imagePath = argv[1];
    int threadsPerBlock = atoi(argv[2]);

    double startAlloc, startAlloc2, endAlloc;
    double startInit, endInit;
    double startConv, endConv;
    double startHD, endHD;
    double startDH, endDH;
    double overhead;

    int img_w; // Ancho
    int img_h; // Alto

    float* h_image = loadPGM32(imagePath, &img_w, &img_h);

    if (h_image == NULL) {
        printf("Error: Image could not be loaded\n");
        return EXIT_FAILURE;
    }

    size_t imageSize = img_w * (size_t)img_h;
    size_t imageSizeBytes = imageSize * sizeof(float);

    int ker_w = 3, ker_h = 3;
    float h_ker[] = {0, -1, 0, -1, 5, -1, 0, -1, 0};
    size_t kernelSizeBytes = ker_w * ker_h * sizeof(float);
    float *d_image, *d_conv, *d_ker;
    float* h_conv = NULL;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    checkCUDAError("Getting device properties");

    startAlloc = get_time();
    startAlloc2 = get_time();

    hipMalloc(&d_image, imageSizeBytes);
    checkCUDAError("Allocating device memory for image");

    hipMalloc(&d_conv, imageSizeBytes);
    checkCUDAError("Allocating device memory for convolution result");

    hipMalloc(&d_ker, kernelSizeBytes);
    checkCUDAError("Allocating device memory for kernel");

    h_conv = (float*)malloc(imageSizeBytes);

    if (h_conv == NULL) {
        printf("Error allocating host memory for convolution result\n");
        return EXIT_FAILURE;
    }

    endAlloc = get_time();

    startHD = get_time();

    hipMemcpy(d_image, h_image, imageSizeBytes, hipMemcpyHostToDevice);
    checkCUDAError("Copying image data to device");

    hipMemcpy(d_ker, h_ker, kernelSizeBytes, hipMemcpyHostToDevice);
    checkCUDAError("Copying kernel data to device");

    endHD = get_time();

    startInit = get_time();

    hipMemset(d_conv, 0, imageSizeBytes);
    checkCUDAError("Initializing convolution result array");

    endInit = get_time();

    int blockDim = (int)sqrt(threadsPerBlock);

    blockDim = (blockDim > 32) ? 32 : blockDim;

    dim3 dimBlock(blockDim, blockDim);

    dim3 dimGrid((img_w + dimBlock.x - 1) / dimBlock.x,
                 (img_h + dimBlock.y - 1) / dimBlock.y);

    if (dimGrid.x > prop.maxGridSize[0]) {
        dimGrid.x = prop.maxGridSize[0]; // Limita al máximo permitido en x
    }

    if (dimGrid.y > prop.maxGridSize[1]) {
        dimGrid.y = prop.maxGridSize[1]; // Limita al máximo permitido en y
    }

    startConv = get_time();

    conv2dKernel<<<dimGrid, dimBlock>>>(d_image, d_conv, img_w, img_h, d_ker,
                                        ker_w, ker_h);

    hipDeviceSynchronize();
    checkCUDAError("Executing convolution kernel");

    endConv = get_time();

    startDH = get_time();

    hipMemcpy(h_conv, d_conv, imageSizeBytes, hipMemcpyDeviceToHost);
    checkCUDAError("Copying convolution results to host");

    endDH = get_time();

    printf("Running sequential version for verification...\n");
    float* h_conv_seq = (float*)malloc(imageSizeBytes);
    if (h_conv_seq == NULL) {
        printf(
            "Error allocating host memory for sequential convolution result\n");
        return EXIT_FAILURE;
    }

    conv2d_sequential(h_image, img_w, img_h, h_conv_seq);

    int errors = 0;
    for (int i = 0; i < imageSize; i++) {
        if (fabs(h_conv[i] - h_conv_seq[i]) > 0.00001) {
            errors++;
        }
    }

    if (errors == 0) {
        printf("Verification successful! Results match.\n");
    } else {
        printf("Verification failed! %d of %ld pixels differ.\n", errors,
               imageSize);
    }

#ifdef DEBUG
    savePGM32("output_sequential.pgm", h_conv_seq, img_w, img_h);
#endif

    int maxBlocksPerSM;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocksPerSM, conv2dKernel,
                                                  blockDim * blockDim, 0);
    checkCUDAError("Calculating occupancy");

    float occupancy = (float)(maxBlocksPerSM * blockDim * blockDim) /
                      prop.maxThreadsPerMultiProcessor;

    overhead = startAlloc2 - startAlloc;
    double alloc_time = endAlloc - startAlloc2 - overhead;
    double init_time = endInit - startInit - overhead;
    double conv_time = endConv - startConv - overhead;
    double hd_time = endHD - startHD - overhead;
    double dh_time = endDH - startDH - overhead;
    double total_time = alloc_time + init_time + conv_time + hd_time + dh_time;

#ifdef DEBUG
    savePGM32("output.pgm", h_conv, img_w, img_h);
#endif

    printf("\nPAE,%s,%d,%d,%d,%f,%.12f,%.12f,%.12f,%.12f,%.12f,%.12f,%.12f,%ld,"
           "PAE\n",
           imagePath, threadsPerBlock, dimGrid.x * dimGrid.y, maxBlocksPerSM,
           occupancy, overhead, alloc_time, init_time, conv_time, hd_time,
           dh_time, total_time, imageSize);

    hipFree(d_image);
    checkCUDAError("Freeing device memory for image");

    hipFree(d_conv);
    checkCUDAError("Freeing device memory for convolution result");

    hipFree(d_ker);
    checkCUDAError("Freeing device memory for kernel");

    free(h_image);
    free(h_conv);
    free(h_conv_seq);

    return EXIT_SUCCESS;
}
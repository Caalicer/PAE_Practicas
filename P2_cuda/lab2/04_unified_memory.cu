/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


// includes, system
#include <stdio.h>
#include <assert.h>

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer and dimension for host memory
    int n, dim = 262144;
    float *h_a, *h_b;

    // pointers for device memory
    float *d_a;

    // allocate unified memory for h_a and d_a;
    hipMallocManaged(  );
    hipMallocManaged(  );
    
    // initialize input data in host
    for (n=0; n<dim; n++)
    {
        h_a[n] = (float) n;
    }

    // copy data from h_a to d_a using a loop in CPU

    // device to host copy h_b = d_a
    hipMemcpy(  );

    // verify the data on the host is correct
    for (n=0; n<dim; n++)
    {
        assert(h_b[n] == h_a[n]);
    }

    // free all memory

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

// nvcc 04_unified_memory.cu - 04_unified_memory
